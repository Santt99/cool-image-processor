#include "hip/hip_runtime.h"
#include "images.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdint.h>
#include <iostream>

using namespace std;

__global__
void blur(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset-x)/width;
    int fsize = 5; // Filter size
    if(offset < width*height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = (offset+ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[offset*3] = output_red/hits;
        output_image[offset*3+1] = output_green/hits;
        output_image[offset*3+2] = output_blue/hits;
        }
}
__global__
void grayscale(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < width*height){
        unsigned char res = (input_image[idx*3] + input_image[idx*3 + 1] + input_image[idx*3+1])/3;
        output_image[idx] = res;
    }
}




void filter (unsigned char* input_image, unsigned char* output_image, int width, int height, int outChannels) {

    unsigned char* dev_input;
    unsigned char* dev_output;
    hipMalloc( (void**) &dev_input, width*height*3*sizeof(unsigned char));
    hipMemcpy( dev_input, input_image, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice );
 
    hipMalloc( (void**) &dev_output, width*height*outChannels*sizeof(unsigned char));
    if (outChannels == 3){
        dim3 blockDims(512,1,1);
        dim3 gridDims((unsigned int) ceil((double)(width*height*3/blockDims.x)), 1, 1 );
        blur<<<gridDims, blockDims>>>(dev_input, dev_output, width, height); 
    }
    else{
        int N = width*height;
        int THREADS_PER_BLOCK = 512;
        grayscale<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(dev_input, dev_output, width, height);
    }


    hipMemcpy(output_image, dev_output, width*height*outChannels*sizeof(unsigned char), hipMemcpyDeviceToHost );

    hipFree(dev_input);
    hipFree(dev_output);

}


